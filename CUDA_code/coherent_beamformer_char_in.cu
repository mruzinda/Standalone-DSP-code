#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
//#include <unistd.h>
#include <hipblas.h>
#include <time.h>
//#include <sys/time.h>
#include <iostream>
#include <string.h>
//#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "coherent_beamformer_char_in.h"

using namespace std;

// Generate simulated data
//signed char* simulate_data();

// Generate simulated weights or coefficients
//float* simulate_coefficients();

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset);

// Convert weights from float to hipComplex
//__global__
//void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex);

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data, int offset);

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power, int offset);

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

//float* h_data = NULL;
//float* h_coeff = NULL;
float* d_data_float = NULL;
signed char* d_data_char = NULL;
hipComplex* d_data_comp = NULL;
float* d_coeff = NULL;
//hipComplex* d_coeff_comp = NULL;
//float* d_coh_bf_out = NULL;
float* d_coh_bf_pow = NULL;
float* h_bf_pow = NULL;
// Allocate memory to all arrays 
void init_beamformer() {
	printf("Here In init_beamformer()! \n");
	// Allocate pinned memory for input data
	//checkCuda(hipHostMalloc((void **)&h_data, N_INPUT * sizeof(float)));
	//printf("Here 1st hipHostMalloc! \n");
	
	// Allocate pinnted memery for beamformer coefficients
	//checkCuda(hipHostMalloc((void **)&h_coeff, N_COEFF * sizeof(float)));
	//printf("Here 2nd hipHostMalloc! \n");
	//hipHostAlloc((void **)&h_data, N_INPUT * sizeof(float));

	// Allocate memory for input data float type
	checkCuda(hipMalloc((void **)&d_data_float, (N_INPUT) * sizeof(float)));
	checkCuda(hipMalloc((void **)&d_data_char, (N_INPUT) * sizeof(signed char)));
	printf("Here 1st hipMalloc! \n");

	// Allocate memory for input data hipComplex type
	checkCuda(hipMalloc((void **)&d_data_comp, (N_INPUT) * sizeof(hipComplex) / 2));
	printf("Here 2nd hipMalloc! \n");

	/*
	size_t f, t;
    	hipSetDevice(0);
    	hipMemGetInfo(&f, &t);
    	fprintf(stdout,"Free: %zu bytes, Available: %zu bytes \n",f,t);
	*/

	// Allocate memory for coefficients float type
	checkCuda(hipMalloc((void **)&d_coeff, N_COEFF * sizeof(float)));
	printf("Here 3rd hipMalloc! \n");

	// Allocate memory for coefficients hipComplex type
	//checkCuda(hipMalloc((void **)&d_coeff_comp, N_COEFF * sizeof(hipComplex) / 2));
	//printf("Here 4th hipMalloc! \n");

	// Allocate memory for coherent beamformer output
	//checkCuda(hipMalloc((void **)&d_coh_bf_out, N_OUTPUT * sizeof(float)));
	//printf("Here 5th hipMalloc! \n");

	// Allocate memory for output power of coherent beamformer
        checkCuda(hipMalloc((void **)&d_coh_bf_pow, (N_BF_POW) * sizeof(float)));
	printf("Here 4th hipMalloc! \n");

	checkCuda(hipHostMalloc((void **)&h_bf_pow, (N_BF_POW) * sizeof(float)));

	return;
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset) {
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index

	// If the input data is not float e.g. signed char, just multiply it by '1.0f' to convert it to a float
	if(f < N_FREQ_STREAM){
		int h_in = data_in_idx(a, p, (f + offset), t);
		int h_tr = data_tr_idx(a, p, (f + offset), t);
		data_tra[h_tr].x = data_in[2*h_in]*1.0f;
		data_tra[h_tr].y = data_in[2*h_in + 1]*1.0f;
	}

	return;
}

/*
// Convert weights from float to hipComplex
__global__
void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex) {
	// Product of antenna and beam dimensions exceeds 1024 so beams
	// are blocks rather than threads to allow for increase in numbers  
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int b = blockIdx.y;  // Beam index
	int f = blockIdx.x;  // Frequency bin index 
	coeff_complex[coeff_idx(a, p, b, f)].x = coeff_float[2*coeff_idx(a, p, b, f)];
	coeff_complex[coeff_idx(a, p, b, f)].y = coeff_float[2*coeff_idx(a, p, b, f) + 1];
	
	return;
}
*/

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data, int offset) {
	/*
	int p = threadIdx.x; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index
	hipComplex bf_product;
	hipComplex bf_in_data;
	hipComplex bf_coeff;
	for (int a = 0; a < N_ANT; a++) { // Antenna index
		bf_in_data.x = input_data[data_tr_idx(a, p, f, t)].x;
		bf_in_data.y = input_data[data_tr_idx(a, p, f, t)].y;
		bf_coeff.x = coeff[coeff_idx(a, p, b, f)].x;
		bf_coeff.y = coeff[coeff_idx(a, p, b, f)].y;
		// Complex multiplication of data and coefficients
		bf_product.x = (bf_in_data.x * bf_coeff.x) - (bf_in_data.y * bf_coeff.y);
		bf_product.y = (bf_in_data.x * bf_coeff.y) + (bf_in_data.y * bf_coeff.x);
		// Beamform (Sum all antennas)
		output_data[2*coh_bf_idx(p, b, f, t)] += bf_product.x;
		output_data[2*coh_bf_idx(p, b, f, t) + 1] += bf_product.y;
	}
	*/
	int a = threadIdx.x; // Antenna index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index

	__shared__ hipFloatComplex reduced_mul[N_ANT];

	if(f < N_FREQ_STREAM){
		for (int p = 0; p < N_POL; p++) { // Polarization index
			// Reinitialize output_data since we are using the input data array to be more efficient
			//int h = coh_bf_idx(p, b, (f + offset), t);
			//output_data[2 * h] = 0;
			//output_data[2 * h + 1] = 0;

	
			int i = data_tr_idx(a, p, (f + offset), t);
			int w = coeff_idx(a, b);

			if (a < N_ANT) {
				reduced_mul[a].x = input_data[i].x * coeff[2*w] + input_data[i].y * coeff[2*w + 1];
				reduced_mul[a].y = input_data[i].y * coeff[2*w] - input_data[i].x * coeff[2*w + 1];
			}
			else {
				reduced_mul[a].x = 0;
				reduced_mul[a].y = 0;
			}
			__syncthreads();

			for (int k = blockDim.x / 2; k > 0; k >>= 1) {
				if (a < k) {
					reduced_mul[a].x += reduced_mul[a + k].x;
					reduced_mul[a].y += reduced_mul[a + k].y;
				}
				__syncthreads();
			}
			if (a == 0) {
				int h1 = coh_bf_idx(p, b, (f + offset), t);
				output_data[2 * h1] += reduced_mul[0].x;
				output_data[2 * h1 + 1] += reduced_mul[0].y;
			}
		
		}
	}
	return;
}

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power, int offset) {
	int b = threadIdx.x; // Beam index
	int f = blockIdx.x;  // Frequency bin index
	int t = blockIdx.y;  // Time sample index

	if(f < N_FREQ_STREAM){	
		// Power = Absolute value squared of output -> r^2 + i^2
		int xp = coh_bf_idx(0, b, (f + offset), t); // X polarization
		int yp = coh_bf_idx(1, b, (f + offset), t); // Y polarization
	
		float x_pol_pow = (bf_volt[2 * xp] * bf_volt[2 * xp]) + (bf_volt[2 * xp + 1] * bf_volt[2 * xp + 1]); // XX*
		float y_pol_pow = (bf_volt[2 * yp] * bf_volt[2 * yp]) + (bf_volt[2 * yp + 1] * bf_volt[2 * yp + 1]); // YY*

		int h = pow_bf_idx(b, (f + offset), t);
		bf_power[h] = x_pol_pow + y_pol_pow; // XX* + YY*
	}
	/*
	bf_power[pow_bf_idx(0, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*xp]) + (bf_volt[2*xp + 1]*bf_volt[2*xp + 1]); // XX*
	bf_power[pow_bf_idx(1, b, f, t)] = (bf_volt[2*yp]*bf_volt[2*yp]) + (bf_volt[2*yp + 1]*bf_volt[2*yp + 1]); // YY*
	bf_power[pow_bf_idx(2, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*yp]) + (bf_volt[2*xp + 1]*bf_volt[2*yp + 1]); // XY* real
	bf_power[pow_bf_idx(3, b, f, t)] = (bf_volt[2*xp + 1]*bf_volt[2*yp]) - (bf_volt[2*xp]*bf_volt[2*yp + 1]); // XY* imag
	*/

	return;
}

// Run beamformer
//void run_beamformer(float* data_in, float* h_coefficient, float* data_out) {
float* run_beamformer(signed char* data_in, float* h_coefficient) {
	/*
	// Allocate input data in pinned memory
	// (This may take longer than it's worth to implement pinned memory)
	*h_data = *data_in;
	*/

	hipError_t err_code;

	//const int freq_chans = N_FREQ_STREAM;
	const int nStreams = N_STREAMS; // Number of streams that make up all of the data. Split in frequency blocks (largest dimension)
	//printf("Total frequency channels: %d , num streams: %d \n", freq_chans, nStreams);
	printf("Total frequency channels: %d , num streams: %d \n", N_FREQ_STREAM, nStreams);

	// Transpose kernel: Specify grid and block dimensions
	dim3 dimBlock_transpose(N_ANT, N_POL, 1);
	dim3 dimGrid_transpose(N_FREQ, N_TIME, 1);

	// Beamformer coefficient kernel (float to complex): Specify grid and block dimensions
	//dim3 dimBlock_bf_coeff(N_ANT, N_POL, 1);
	//dim3 dimGrid_bf_coeff(N_FREQ, N_BEAM, 1);

	// Coherent beamformer kernel: Specify grid and block dimensions
	//dim3 dimBlock_coh_bf(N_ANT, N_POL, 1);
	dim3 dimBlock_coh_bf(N_ANT, 1, 1);
	dim3 dimGrid_coh_bf(N_FREQ, N_TIME, N_BEAM);

	// Output power of beamformer kernel: Specify grid and block dimensions
	dim3 dimBlock_bf_pow(N_BEAM, 1, 1);
	dim3 dimGrid_bf_pow(N_FREQ, N_TIME, 1);

	float* d_data_bf = d_data_float;
	signed char* d_data_in = d_data_char;
	hipComplex* d_data_tra = d_data_comp;
	float* d_coefficient = d_coeff;
	//hipComplex* d_coeff_c = d_coeff_comp;
	//float* d_bf_output = d_coh_bf_out;
	float* d_bf_pow = d_coh_bf_pow;
	float* data_out = h_bf_pow;
        
	printf("Before hipMemcpy(HtoD) coefficients! \n");
	// Copy beamformer coefficients from host to device
	checkCuda(hipMemcpy(d_coefficient, h_coefficient, N_COEFF * sizeof(float), hipMemcpyHostToDevice));
	printf("Here hipMemcpy(HtoD) coefficients! \n");

	// CUDA streams and events applied for optimization to possibly eliminate stalls.
	// hipMemcpy(HtoD) and data_restructure kernel	
	const int streamSizeIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM);
	//const int streamSizeIn = freq_chans;
	//const unsigned long int streamBytesIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM*sizeof(float));
	const unsigned long int streamBytesIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM*sizeof(signed char));
	printf("Input size: %d in bytes: %lu \n", streamSizeIn, streamBytesIn);

	// hipMemcpy(HtoD) coefficients
	//const int streamSizeCo = N_ANT*N_BEAM*N_TIME/nStreams;
	//const int streamBytesCo = streamSizeCo * sizeof(float);

	// coherent_beamformer kernel  
	//const unsigned int streamSizeBF = (unsigned int)(2*N_BEAM*N_POL*N_TIME*N_FREQ_STREAM);
	//const unsigned int streamBytesBF = streamSizeBF * sizeof(float);
	//printf("BF output size: %u \n", streamSizeBF);

	// beamformer_power kernel and hipMemcpy(DtoH)
	const int streamSizePow = (N_BEAM*N_TIME*N_FREQ_STREAM);
	//const int streamSizePow = freq_chans;
	const unsigned long int streamBytesPow = (N_BEAM*N_TIME*N_FREQ_STREAM*sizeof(float));	
	printf("BF power output size: %d in bytes: %lu \n", streamSizePow, streamBytesPow);

	// Create events and streams
	// Events ////////////////////////////////////
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));		
	checkCuda(hipEventRecord(startEvent, 0));
	/////////////////////////////////////////////

	hipStream_t stream[nStreams];

	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamCreate(&stream[i]));
	}

	for (int i = 0; i < nStreams; ++i){

		int offset = i * N_FREQ_STREAM;
		int offset_in = i * streamSizeIn;
		// Copy input data from host to device
		checkCuda(hipMemcpyAsync(&d_data_in[offset_in], &data_in[offset_in], streamBytesIn, hipMemcpyHostToDevice, stream[i]));
		//checkCuda(hipMemcpyAsync(d_data_in, &data_in[offset_in], streamBytesIn, hipMemcpyHostToDevice, stream[i]));
		printf("First hipMemcpyAsync(HtoD) in run_beamformer(), offset_in = %d and offset = %d \n", offset_in, offset);
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Perform transpose on the data and convert to floats  
		data_transpose<<<dimGrid_transpose, dimBlock_transpose, 0, stream[i]>>>(d_data_in, d_data_tra, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: data_transpose() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		printf("Here data_transpose! \n");
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Perform beamforming operation
		// Use d_data_in for output since it is no longer being utilized,
		// and it is the same size as the output (4 GiB).
		//unsigned long int offset_bf = i * streamSizeBF;
		coherent_beamformer<<<dimGrid_coh_bf, dimBlock_coh_bf, 0, stream[i]>>>(d_data_tra, d_coefficient, d_data_bf, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: coherent_beamformer() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		//printf("Here coherent_beamformer, offset = %d \n", offset);	
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Compute power of beamformer output (abs()^2)
		int offset_pow = i * streamSizePow;
		beamformer_power<<<dimGrid_bf_pow, dimBlock_bf_pow, 0, stream[i]>>>(d_data_bf, d_bf_pow, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: beamformer_power() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		printf("Here beamformer_power, offset_pow = %d and offset = %d \n", offset_pow, offset);
	}
	for (int i = 0; i < nStreams; ++i){
		int offset_pow = i * streamSizePow;
		// Copy output power from device to host
		checkCuda(hipMemcpyAsync(&data_out[offset_pow], &d_bf_pow[offset_pow], streamBytesPow, hipMemcpyDeviceToHost, stream[i]));
		//checkCuda(hipMemcpyAsync(&data_out[offset_pow], d_bf_pow, streamBytesPow, hipMemcpyDeviceToHost, stream[i]));
		printf("Here hipMemcpyAsync(DtoH)! \n");

	}

	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamSynchronize(stream[i]));
	}

	// Events ////////////////////////////////////
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	/////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////
	// Clean up streams
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamDestroy(stream[i]));
	}

	return data_out;
}

// Generate simulated data
signed char* simulate_data() {
	signed char* data_sim;
	data_sim = (signed char*)calloc(N_INPUT, sizeof(signed char));
	//checkCuda(hipHostMalloc((void **)&data_sim, N_INPUT * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Repeating sequence of 1 to 64
	sim_flag = 2 -> Sequence of 1 to 64 placed in a particular bin (bin 6 for now)
	sim flag = 3 -> Simulated radio source in center beam assuming ULA
	*/
	int sim_flag = 1;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_INPUT / 2); i++) {
			data_sim[2 * i] = 1;
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int f = 0; f < N_FREQ; f++) {
					for (int a = 0; a < N_ANT; a++) {
						if (tmp >= N_ANT) {
							tmp = 0;
						}
						tmp = (tmp + 1) % (N_ANT+1);
						data_sim[2 * data_in_idx(a, p, f, t)] = tmp;
					}
				}
			}
		}
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int a = 0; a < N_ANT; a++) {
					if (tmp >= N_ANT) {
						tmp = 0;
					}
					tmp = (tmp + 1) % (N_ANT+1);
					data_sim[2 * data_in_idx(a, p, 5, t)] = tmp;
					data_sim[2 * data_in_idx(a, p, 2, t)] = tmp;
				}
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		float chan_band = 1; // Fine channel bandwidth in Hz

		//float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		//for (int i = 0; i < N_FREQ; i++) {
		//	rf_freqs[i] = chan_band * i + c_freq;
		//}

		//float* theta = (float*)calloc(N_TIME, sizeof(float)); // SOI direction/angle of arrival
		//float* tau = (float*)calloc(N_TIME, sizeof(float)); // Delay

		float theta = 0; // SOI direction/angle of arrival
		float tau = 0; // Delay
		float rf_freqs = 0;

		for (int t = 0; t < N_TIME; t++) {
			theta = (t - (N_TIME / 2)) + 90; // SOI direction/angle of arrival -> Moving across array over time i.e. angle changes each time sample
			tau = d * cos(theta) / c; // Delay
			for (int f = 0; f < N_FREQ; f++) {
				rf_freqs = chan_band * f + c_freq;
				for (int a = 0; a < N_ANT; a++) {
					// X polarization
					data_sim[2 * data_in_idx(a, 0, f, t)] = cos(2 * PI * rf_freqs * a * tau);
					data_sim[2 * data_in_idx(a, 0, f, t) + 1] = sin(2 * PI * rf_freqs * a * tau);
					// Y polarization
					data_sim[2 * data_in_idx(a, 1, f, t)] = cos(2 * PI * rf_freqs * a * tau);
					data_sim[2 * data_in_idx(a, 1, f, t) + 1] = sin(2 * PI * rf_freqs * a * tau); // Make this negative if a different polarization is tested
				}
			}
		}
	}
	//for (int a = 0; a < N_ANT; a++){
	//	printf("Antenna %d = %f\n", (a+1), data_sim[2 * data_in_idx(a, 0, 0, 0)]);
	//}
	return data_sim;
}

// Generate simulated weights or coefficients
float* simulate_coefficients() {
	float* coeff_sim;
	coeff_sim = (float*)calloc(N_COEFF, sizeof(float));
	//checkCuda(hipHostMalloc((void **)&coeff_sim, N_COEFF * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Scale each beam by incrementing value i.e. beam 1 = 1, beam 2 = 2, ..., beam 64 = 64
	sim_flag = 2 -> Scale each beam by incrementing value in a particular bin (bin 3 and 6 for now). Match simulated data sim_flag = 2
	sim flag = 3 -> Simulated beams from 58 to 122 degrees. Assuming a ULA.
	*/
	int sim_flag = 0;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_COEFF / 2); i++) {
			coeff_sim[2 * i] = 1;
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
		
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		//float chan_band = 1.59; // Fine channel bandwidth in Hz

		//float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		//for (int i = 0; i < N_FREQ; i++) {
		//	rf_freqs[i] = chan_band * i + c_freq;
		//}

		//float* theta = (float*)calloc(N_TIME, sizeof(float)); // Beam angle from 58 to 122 degrees
		//float* tau_beam = (float*)calloc(N_BEAM, sizeof(float)); // Delay

		float theta = 0; // Beam angle from 58 to 122 degrees
		float tau_beam = 0; // Delay

		for (int b = 0; b < N_BEAM; b++) {
			theta = (b - (N_BEAM / 2)) + 90; // Beam angle from 58 to 122 degrees - Given SOI at 90 deg or moving across array, the beam with the most power is beamm 33
			tau_beam = d * cos(theta) / c; // Delay
			for (int a = 0; a < N_ANT; a++) {
				coeff_sim[2 * coeff_idx(a, b)] = cos(2 * PI * c_freq * a * tau_beam);
				coeff_sim[2 * coeff_idx(a, b) + 1] = sin(2 * PI * c_freq * a * tau_beam);
			}
		}
	}
	//for (int b = 0; b < N_BEAM; b++){
	//	printf("Beam %d = %f\n", (b+1), coeff_sim[2 * coeff_idx(0, b)]);
	//}

	return coeff_sim;
}

// The input_data_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
//void input_data_pin(float * data_in_pin) {
void input_data_pin(signed char * data_in_pin) {
	checkCuda(hipHostRegister(data_in_pin, N_INPUT*sizeof(signed char), hipHostRegisterPortable));
}

// The data_coeff_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
void coeff_pin(float * data_coeff_pin) {
	checkCuda(hipHostRegister(data_coeff_pin, N_COEFF*sizeof(float), hipHostRegisterPortable));
}

// The output_data_pin() function uses hipHostRegister() to allocate the output host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
void output_data_pin(float * data_out_pin) {
	checkCuda(hipHostRegister(data_out_pin, N_BF_POW*sizeof(float), hipHostRegisterPortable));
}

// The coefficient_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
//void coefficient_pin(float * coeff_pin) {
//	checkCuda(hipHostRegister(coeff_pin, N_COEFF*sizeof(float), hipHostRegisterPortable));
//}

// Unregister host arrays from pinned memory
void unregister_data(void * data_unregister){
	checkCuda(hipHostUnregister(data_unregister));
}

// Free memory
void cohbfCleanup() {
	// Free up GPU memory at the end of a program
	//if (h_data != NULL) {
	//	hipHostFree(h_data);
	//}
	if (d_data_char != NULL) {
		hipFree(d_data_char);
	}
	if (d_data_float != NULL) {
		hipFree(d_data_float);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
	if (d_coeff != NULL) {
		hipFree(d_coeff);
	}
	//if (d_coeff_comp != NULL) {
	//	hipFree(d_coeff_comp);
	//}
	//if (d_coh_bf_out != NULL) {
	//	hipFree(d_coh_bf_out);
	//}
	if (d_coh_bf_pow != NULL) {
		hipFree(d_coh_bf_pow);
	}
}


// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main() {
	printf("Here!\n");

	// Allocate memory to all arrays used by run_beamformer() 
	init_beamformer();


	printf("Here1!\n");
	// Generate simulated data
	signed char* sim_data = simulate_data();
	// Register the array in pinned memory to speed HtoD mem copy
	input_data_pin(sim_data);

	printf("Here2!\n");

	// Generate simulated weights or coefficients
	float* sim_coefficients = simulate_coefficients();
	//printf("Here3!\n");
	// Register the array in pinned memory to speed HtoD mem copy
	coeff_pin(sim_coefficients);

	printf("Here4!\n");

	// Allocate memory for output array
	float* output_data;
	//output_data = (float*)calloc(N_BF_POW, sizeof(float));
	//output_data = (float*)calloc(N_OUTPUT, sizeof(float));
	//output_data_pin(output_data);

	printf("Here5!\n");

	// Run beamformer 
	output_data = run_beamformer(sim_data, sim_coefficients);
	//run_beamformer(h_data, h_coeff, output_data);

	printf("Here6, Beamformer output: %f \n", output_data[0]);
	
	// Write data to text file for analysis
	char output_filename[128];

	printf("Here7!\n");

	//strcpy(output_filename, "C:\Users\ruzie\OneDrive\Desktop\Work\CUDA_code\output_d.txt");
	strcpy(output_filename, "output_d_cuda.txt");

	printf("Here8!\n");

	FILE* output_file;

	printf("Here9!\n");

	output_file = fopen(output_filename, "w");

	printf("Here10!\n");

	for (int ii = 0; ii < N_BF_POW; ii++) {
		fprintf(output_file, "%g\n", output_data[ii]);
	}

	printf("Here11!\n");

	fclose(output_file);

	printf("Closed output file.\n");

	//unregister_data((float *)sim_data);
	//free(sim_data);
	printf("After unregister.\n");	
	//free(sim_coefficients);
	printf("After freeing coefficients.\n");
	//free(output_data);	

	printf("Freed output array and unregistered arrays in pinned memory.\n");

	// Free up device memory
	//hipHostFree(h_data);
	//hipHostFree(h_coeff);
	cohbfCleanup();

	printf("Here11!\n");

	return 0;
}
