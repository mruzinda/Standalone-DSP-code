#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
//#include <unistd.h>
#include <hipblas.h>
#include <time.h>
#include <sys/time.h>
#include <iostream>
#include <string.h>
//#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "coherent_beamformer_char_in.h"

using namespace std;

// Generate simulated data
//signed char* simulate_data();

// Generate simulated weights or coefficients
//float* simulate_coefficients();

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset);

// Convert weights from float to hipComplex
//__global__
//void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex);

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data, int offset);

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power, int offset);
//void beamformer_power(float* bf_volt, signed char* bf_power, int offset);


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

//float* h_data = NULL;
//float* h_coeff = NULL;
float* d_data_float = NULL;
signed char* d_data_char = NULL;
hipComplex* d_data_comp = NULL;
float* d_coeff = NULL;
//hipComplex* d_coeff_comp = NULL;
//float* d_coh_bf_out = NULL;
float* d_coh_bf_pow = NULL;
float* h_bf_pow = NULL;
//signed char* d_coh_bf_pow = NULL;
//signed char* h_bf_pow = NULL;
// Allocate memory to all arrays 
void init_beamformer() {
	printf("Here In init_beamformer()! \n");
	// Allocate pinned memory for input data
	//checkCuda(hipHostMalloc((void **)&h_data, N_INPUT * sizeof(float)));
	//printf("Here 1st hipHostMalloc! \n");
	
	// Allocate pinnted memery for beamformer coefficients
	//checkCuda(hipHostMalloc((void **)&h_coeff, N_COEFF * sizeof(float)));
	//printf("Here 2nd hipHostMalloc! \n");
	//hipHostAlloc((void **)&h_data, N_INPUT * sizeof(float));

	// Allocate memory for input data float type
	checkCuda(hipMalloc((void **)&d_data_float, (N_INPUT) * sizeof(float)));
	checkCuda(hipMalloc((void **)&d_data_char, (N_INPUT) * sizeof(signed char)));
	printf("Here 1st hipMalloc! \n");

	// Allocate memory for input data hipComplex type
	checkCuda(hipMalloc((void **)&d_data_comp, (N_INPUT) * sizeof(hipComplex) / 2));
	printf("Here 2nd hipMalloc! \n");

	/*
	size_t f, t;
    	hipSetDevice(0);
    	hipMemGetInfo(&f, &t);
    	fprintf(stdout,"Free: %zu bytes, Available: %zu bytes \n",f,t);
	*/

	// Allocate memory for coefficients float type
	checkCuda(hipMalloc((void **)&d_coeff, N_COEFF * sizeof(float)));
	printf("Here 3rd hipMalloc! \n");

	// Allocate memory for coefficients hipComplex type
	//checkCuda(hipMalloc((void **)&d_coeff_comp, N_COEFF * sizeof(hipComplex) / 2));
	//printf("Here 4th hipMalloc! \n");

	// Allocate memory for coherent beamformer output
	//checkCuda(hipMalloc((void **)&d_coh_bf_out, N_OUTPUT * sizeof(float)));
	//printf("Here 5th hipMalloc! \n");

	// Allocate memory for output power of coherent beamformer
        checkCuda(hipMalloc((void **)&d_coh_bf_pow, (N_BF_POW) * sizeof(float)));
	printf("Here 4th hipMalloc! \n");

	checkCuda(hipHostMalloc((void **)&h_bf_pow, (N_BF_POW) * sizeof(float)));

	//checkCuda(hipMalloc((void **)&d_coh_bf_pow, (N_BF_POW) * sizeof(signed char)));
	//printf("Here 4th hipMalloc! \n");

	//checkCuda(hipHostMalloc((void **)&h_bf_pow, (N_BF_POW) * sizeof(signed char)));

	return;
}

// Set arrays to zero after a block is processed
void set_to_zero(){
	checkCuda(hipMemset(d_data_float, 0, (N_INPUT) * sizeof(float)));
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(signed char* data_in, hipComplex* data_tra, int offset) {
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int f = blockIdx.y;  // Frequency index
	int t = blockIdx.x;  // Time sample index

	// If the input data is not float e.g. signed char, just multiply it by '1.0f' to convert it to a float
	if(f < N_FREQ_STREAM){
		int h_in = data_in_idx(a, p, (f + offset), t);
		int h_tr = data_tr_idx(a, p, (f + offset), t);
		//if(a < N_REAL_ANT){
		data_tra[h_tr].x = data_in[2*h_in]*1.0f;
		data_tra[h_tr].y = data_in[2*h_in + 1]*1.0f;
		//}else{
		//	data_tra[h_tr].x = 0*1.0f;
		//	data_tra[h_tr].y = 0*1.0f;
		//}
	}

	return;
}

/*
// Convert weights from float to hipComplex
__global__
void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex) {
	// Product of antenna and beam dimensions exceeds 1024 so beams
	// are blocks rather than threads to allow for increase in numbers  
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int b = blockIdx.y;  // Beam index
	int f = blockIdx.x;  // Frequency bin index 
	coeff_complex[coeff_idx(a, p, b, f)].x = coeff_float[2*coeff_idx(a, p, b, f)];
	coeff_complex[coeff_idx(a, p, b, f)].y = coeff_float[2*coeff_idx(a, p, b, f) + 1];
	
	return;
}
*/

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data, int offset) {
	/*
	int p = threadIdx.x; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index
	hipComplex bf_product;
	hipComplex bf_in_data;
	hipComplex bf_coeff;
	for (int a = 0; a < N_ANT; a++) { // Antenna index
		bf_in_data.x = input_data[data_tr_idx(a, p, f, t)].x;
		bf_in_data.y = input_data[data_tr_idx(a, p, f, t)].y;
		bf_coeff.x = coeff[coeff_idx(a, p, b, f)].x;
		bf_coeff.y = coeff[coeff_idx(a, p, b, f)].y;
		// Complex multiplication of data and coefficients
		bf_product.x = (bf_in_data.x * bf_coeff.x) - (bf_in_data.y * bf_coeff.y);
		bf_product.y = (bf_in_data.x * bf_coeff.y) + (bf_in_data.y * bf_coeff.x);
		// Beamform (Sum all antennas)
		output_data[2*coh_bf_idx(p, b, f, t)] += bf_product.x;
		output_data[2*coh_bf_idx(p, b, f, t) + 1] += bf_product.y;
	}
	*/
	int a = threadIdx.x; // Antenna index
	int f = blockIdx.y;  // Frequency index
	int t = blockIdx.x;  // Time sample index
	int b = blockIdx.z;  // Beam index

	__shared__ hipFloatComplex reduced_mul[N_ANT];

	if(f < N_FREQ_STREAM){
		for (int p = 0; p < N_POL; p++) { // Polarization index
			// Reinitialize output_data since we are using the input data array to be more efficient
			//int h = coh_bf_idx(p, b, (f + offset), t);
			//output_data[2 * h] = 0;
			//output_data[2 * h + 1] = 0;

	
			int i = data_tr_idx(a, p, (f + offset), t);
			int w = coeff_idx(a, b);

			if (a < N_ANT) {
				reduced_mul[a].x = input_data[i].x * coeff[2*w] + input_data[i].y * coeff[2*w + 1];
				reduced_mul[a].y = input_data[i].y * coeff[2*w] - input_data[i].x * coeff[2*w + 1];
			}
			else {
				reduced_mul[a].x = 0;
				reduced_mul[a].y = 0;
			}
			__syncthreads();

			for (int k = blockDim.x / 2; k > 0; k >>= 1) {
				if (a < k) {
					reduced_mul[a].x += reduced_mul[a + k].x;
					reduced_mul[a].y += reduced_mul[a + k].y;
				}
				__syncthreads();
			}
			if (a == 0) {
				int h1 = coh_bf_idx(p, b, (f + offset), t);
				output_data[2 * h1] += reduced_mul[0].x;
				output_data[2 * h1 + 1] += reduced_mul[0].y;
			}
		
		}
	}
	return;
}


// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power, int offset) {
	int b = threadIdx.x; // Beam index
	int f = blockIdx.y;  // Frequency bin index
	int t = blockIdx.x;  // Time sample index

	if(f < N_FREQ_STREAM){	
		// Power = Absolute value squared of output -> r^2 + i^2
		int xp = coh_bf_idx(0, b, (f + offset), t); // X polarization
		int yp = coh_bf_idx(1, b, (f + offset), t); // Y polarization
		
		float x_pol_pow = (bf_volt[2 * xp] * bf_volt[2 * xp]) + (bf_volt[2 * xp + 1] * bf_volt[2 * xp + 1]); // XX*
		float y_pol_pow = (bf_volt[2 * yp] * bf_volt[2 * yp]) + (bf_volt[2 * yp + 1] * bf_volt[2 * yp + 1]); // YY*

		int h = pow_bf_idx(b, (f + offset), t);
		bf_power[h] = x_pol_pow + y_pol_pow; // XX* + YY*
	}
	
	//bf_power[pow_bf_idx(0, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*xp]) + (bf_volt[2*xp + 1]*bf_volt[2*xp + 1]); // XX*
	//bf_power[pow_bf_idx(1, b, f, t)] = (bf_volt[2*yp]*bf_volt[2*yp]) + (bf_volt[2*yp + 1]*bf_volt[2*yp + 1]); // YY*
	//bf_power[pow_bf_idx(2, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*yp]) + (bf_volt[2*xp + 1]*bf_volt[2*yp + 1]); // XY* real
	//bf_power[pow_bf_idx(3, b, f, t)] = (bf_volt[2*xp + 1]*bf_volt[2*yp]) - (bf_volt[2*xp]*bf_volt[2*yp + 1]); // XY* imag
	
	return;
}

/*
// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, signed char* bf_power, int offset) {
//void beamformer_power(float* bf_volt, float* bf_power, int offset) {
	int b = threadIdx.x; // Beam index
	int f = blockIdx.x;  // Frequency bin index
	int t = blockIdx.y;  // Time sample index

	if(f < N_FREQ_STREAM){	
		// Power = Absolute value squared of output -> r^2 + i^2
		int xp = coh_bf_idx(0, b, (f + offset), t); // X polarization
		int yp = coh_bf_idx(1, b, (f + offset), t); // Y polarization
		
		float x_pol_pow = (bf_volt[2 * xp] * bf_volt[2 * xp]) + (bf_volt[2 * xp + 1] * bf_volt[2 * xp + 1]); // XX*
		float y_pol_pow = (bf_volt[2 * yp] * bf_volt[2 * yp]) + (bf_volt[2 * yp + 1] * bf_volt[2 * yp + 1]); // YY*

		float tmp_pow = x_pol_pow + y_pol_pow; // XX* + YY*
		// int h = pow_bf_idx(b, (f + offset), t);
		// bf_power[h] = x_pol_pow + y_pol_pow; // XX* + YY*

		int xi = coh_bf_idx(0, 0, 0, 0);
		int yi = coh_bf_idx(1, 0, 0, 0);
		float tmp_x_pol_pow = (bf_volt[2 * xi] * bf_volt[2 * xi]) + (bf_volt[2 * xi + 1] * bf_volt[2 * xi + 1]); // XX*
		float tmp_y_pol_pow = (bf_volt[2 * yi] * bf_volt[2 * yi]) + (bf_volt[2 * yi + 1] * bf_volt[2 * yi + 1]); // YY*
		float tmp_pow_init = tmp_x_pol_pow + tmp_y_pol_pow;


		// Find minimum value of total power
		float min_pow = tmp_pow_init;
		min_pow = min(min_pow, tmp_pow);

		// Find maximum value of total power
		float max_pow = tmp_pow_init;
		max_pow = max(max_pow, tmp_pow);

		// Requantize output from 32 bit floats to 8 bit signed chars
		int h = pow_bf_idx(b, (f + offset), t);
		bf_power[h] = (signed char)((((tmp_pow - min_pow)/(max_pow - min_pow)) - 0.5)*256); // XX* + YY*
	}
	
	
	//bf_power[pow_bf_idx(0, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*xp]) + (bf_volt[2*xp + 1]*bf_volt[2*xp + 1]); // XX*
	//bf_power[pow_bf_idx(1, b, f, t)] = (bf_volt[2*yp]*bf_volt[2*yp]) + (bf_volt[2*yp + 1]*bf_volt[2*yp + 1]); // YY*
	//bf_power[pow_bf_idx(2, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*yp]) + (bf_volt[2*xp + 1]*bf_volt[2*yp + 1]); // XY* real
	//bf_power[pow_bf_idx(3, b, f, t)] = (bf_volt[2*xp + 1]*bf_volt[2*yp]) - (bf_volt[2*xp]*bf_volt[2*yp + 1]); // XY* imag
	

	return;
}
*/
// Run beamformer
//void run_beamformer(float* data_in, float* h_coefficient, float* data_out) {
float* run_beamformer(signed char* data_in, float* h_coefficient) {
//signed char* run_beamformer(signed char* data_in, float* h_coefficient) {
	/*
	// Allocate input data in pinned memory
	// (This may take longer than it's worth to implement pinned memory)
	*h_data = *data_in;
	*/

	hipError_t err_code;

	//const int freq_chans = N_FREQ_STREAM;
	const int nStreams = N_STREAMS; // Number of streams that make up all of the data. Split in frequency blocks (largest dimension)
	//printf("Total frequency channels: %d , num streams: %d \n", freq_chans, nStreams);
	//printf("Total frequency channels: %d , num streams: %d \n", N_FREQ_STREAM, nStreams);

	// Transpose kernel: Specify grid and block dimensions
	dim3 dimBlock_transpose(N_ANT, N_POL, 1);
	dim3 dimGrid_transpose(N_TIME, N_FREQ, 1);
	//dim3 dimGrid_transpose(N_FREQ, N_TIME, 1);

	// Beamformer coefficient kernel (float to complex): Specify grid and block dimensions
	//dim3 dimBlock_bf_coeff(N_ANT, N_POL, 1);
	//dim3 dimGrid_bf_coeff(N_FREQ, N_BEAM, 1);

	// Coherent beamformer kernel: Specify grid and block dimensions
	//dim3 dimBlock_coh_bf(N_ANT, N_POL, 1);
	dim3 dimBlock_coh_bf(N_ANT, 1, 1);
	dim3 dimGrid_coh_bf(N_TIME, N_FREQ, N_BEAM);

	// Output power of beamformer kernel: Specify grid and block dimensions
	dim3 dimBlock_bf_pow(N_BEAM, 1, 1);
	dim3 dimGrid_bf_pow(N_TIME, N_FREQ, 1);

	float* d_data_bf = d_data_float;
	signed char* d_data_in = d_data_char;
	hipComplex* d_data_tra = d_data_comp;
	float* d_coefficient = d_coeff;
	//hipComplex* d_coeff_c = d_coeff_comp;
	//float* d_bf_output = d_coh_bf_out;
	float* d_bf_pow = d_coh_bf_pow;
	float* data_out = h_bf_pow;

	//signed char* d_bf_pow = d_coh_bf_pow;
	//signed char* data_out = h_bf_pow;

	//printf("Before hipMemcpy(HtoD) coefficients! \n");
	// Copy beamformer coefficients from host to device
	checkCuda(hipMemcpy(d_coefficient, h_coefficient, N_COEFF * sizeof(float), hipMemcpyHostToDevice));
	//printf("Here hipMemcpy(HtoD) coefficients! \n");

	// CUDA streams and events applied for optimization to possibly eliminate stalls.
	// hipMemcpy(HtoD) and data_restructure kernel	
	const int streamSizeIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM);
	//const int streamSizeIn = freq_chans;
	//const unsigned long int streamBytesIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM*sizeof(float));
	const unsigned long int streamBytesIn = (2*N_ANT*N_POL*N_TIME*N_FREQ_STREAM*sizeof(signed char));
	//printf("Input size: %d in bytes: %lu \n", streamSizeIn, streamBytesIn);

	// hipMemcpy(HtoD) coefficients
	//const int streamSizeCo = N_ANT*N_BEAM*N_TIME/nStreams;
	//const int streamBytesCo = streamSizeCo * sizeof(float);

	// coherent_beamformer kernel  
	//const unsigned int streamSizeBF = (unsigned int)(2*N_BEAM*N_POL*N_TIME*N_FREQ_STREAM);
	//const unsigned int streamBytesBF = streamSizeBF * sizeof(float);
	//printf("BF output size: %u \n", streamSizeBF);

	// beamformer_power kernel and hipMemcpy(DtoH)
	const int streamSizePow = (N_BEAM*N_TIME*N_FREQ_STREAM);
	//const int streamSizePow = freq_chans;
	const unsigned long int streamBytesPow = (N_BEAM*N_TIME*N_FREQ_STREAM*sizeof(float));
	//const unsigned long int streamBytesPow = (N_BEAM*N_TIME*N_FREQ_STREAM*sizeof(signed char));	
	//printf("BF power output size: %d in bytes: %lu \n", streamSizePow, streamBytesPow);

	// Create events and streams
	// Events ////////////////////////////////////
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));		
	checkCuda(hipEventRecord(startEvent, 0));
	/////////////////////////////////////////////

	hipStream_t stream[nStreams];

	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamCreate(&stream[i]));
	}

	for (int i = 0; i < nStreams; ++i){

		//int offset = i * N_FREQ_STREAM;
		int offset_in = i * streamSizeIn;
		// Copy input data from host to device
		checkCuda(hipMemcpyAsync(&d_data_in[offset_in], &data_in[offset_in], streamBytesIn, hipMemcpyHostToDevice, stream[i]));
		//checkCuda(hipMemcpyAsync(d_data_in, &data_in[offset_in], streamBytesIn, hipMemcpyHostToDevice, stream[i]));
		//printf("First hipMemcpyAsync(HtoD) in run_beamformer(), offset_in = %d and offset = %d \n", offset_in, offset);
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Perform transpose on the data and convert to floats  
		data_transpose<<<dimGrid_transpose, dimBlock_transpose, 0, stream[i]>>>(d_data_in, d_data_tra, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: data_transpose() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		//printf("Here data_transpose! \n");
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Perform beamforming operation
		// Use d_data_in for output since it is no longer being utilized,
		// and it is the same size as the output (4 GiB).
		//unsigned long int offset_bf = i * streamSizeBF;
		coherent_beamformer<<<dimGrid_coh_bf, dimBlock_coh_bf, 0, stream[i]>>>(d_data_tra, d_coefficient, d_data_bf, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: coherent_beamformer() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		//printf("Here coherent_beamformer, offset = %d \n", offset);	
	}
	for (int i = 0; i < nStreams; ++i){
		int offset = i * N_FREQ_STREAM;
		// Compute power of beamformer output (abs()^2)
		//int offset_pow = i * streamSizePow;
		beamformer_power<<<dimGrid_bf_pow, dimBlock_bf_pow, 0, stream[i]>>>(d_data_bf, d_bf_pow, offset);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: beamformer_power() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		//printf("Here beamformer_power, offset_pow = %d and offset = %d \n", offset_pow, offset);
	}
	for (int i = 0; i < nStreams; ++i){
		int offset_pow = i * streamSizePow;
		// Copy output power from device to host
		checkCuda(hipMemcpyAsync(&data_out[offset_pow], &d_bf_pow[offset_pow], streamBytesPow, hipMemcpyDeviceToHost, stream[i]));
		//checkCuda(hipMemcpyAsync(&data_out[offset_pow], d_bf_pow, streamBytesPow, hipMemcpyDeviceToHost, stream[i]));
		//printf("Here hipMemcpyAsync(DtoH)! \n");

	}

	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamSynchronize(stream[i]));
	}

	// Events ////////////////////////////////////
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	/////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////
	// Clean up streams
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamDestroy(stream[i]));
	}

	return data_out;
}

// Generate simulated data
signed char* simulate_data() {
	signed char* data_sim;
	data_sim = (signed char*)calloc(N_INPUT, sizeof(signed char));
	//checkCuda(hipHostMalloc((void **)&data_sim, N_INPUT * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Repeating sequence of 1 to 64
	sim_flag = 2 -> Sequence of 1 to 64 placed in a particular bin (bin 6 for now)
	sim flag = 3 -> Simulated radio source in center beam assuming ULA
	*/
	int sim_flag = 3;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_INPUT / 2); i++) {
			if(i < (N_REAL_INPUT/2)){
				data_sim[2 * i] = 1;
			}else{
				data_sim[2 * i] = 0;
			}
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int f = 0; f < N_FREQ; f++) {
					for (int a = 0; a < N_ANT; a++) {
						if (tmp >= N_ANT) {
							tmp = 0;
						}
						tmp = (tmp + 1) % (N_ANT+1);
						if(a < N_REAL_ANT){
							data_sim[2 * data_in_idx(a, p, f, t)] = tmp;
						}else{
							data_sim[2 * data_in_idx(a, p, f, t)] = 0;
						}
					}
				}
			}
		}
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int a = 0; a < N_ANT; a++) {
					if (tmp >= N_ANT) {
						tmp = 0;
					}
					tmp = (tmp + 1) % (N_ANT+1);
					if(a < N_REAL_ANT){
						data_sim[2 * data_in_idx(a, p, 5, t)] = tmp;
						data_sim[2 * data_in_idx(a, p, 2, t)] = tmp;
					}else{
						data_sim[2 * data_in_idx(a, p, 5, t)] = 0;
						data_sim[2 * data_in_idx(a, p, 2, t)] = 0;
					}
				}
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		float chan_band = 1; // Fine channel bandwidth in Hz

		//float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		//for (int i = 0; i < N_FREQ; i++) {
		//	rf_freqs[i] = chan_band * i + c_freq;
		//}

		//float* theta = (float*)calloc(N_TIME, sizeof(float)); // SOI direction/angle of arrival
		//float* tau = (float*)calloc(N_TIME, sizeof(float)); // Delay

		double theta = 0; // SOI direction/angle of arrival
		double tau = 0; // Delay
		double rf_freqs = 0;
		double cb = 90; // Center beam in degrees

		float tmp_max = 1.0;
		float tmp_min = -1.0;

		for (int t = 0; t < N_TIME; t++) {
			// Reduce the range of angles in order to prevent wrap around - That's what the 100 and 200 are for.
			theta = ((t/50 - (N_TIME / 100)) + cb)*PI/180; // SOI direction/angle of arrival -> Moving across array over time i.e. angle changes each time sample
			tau = d * cos(theta) / c; // Delay
			for (int f = 0; f < N_FREQ; f++) {
				rf_freqs = chan_band * f + c_freq;
				for (int a = 0; a < N_ANT; a++) {
					if(a < N_REAL_ANT){
						// Requantize from doubles/floats to signed chars with a range from -128 to 127
						// X polarization
						data_sim[2 * data_in_idx(a, 0, f, t)] = (signed char)((((cos(2 * PI * rf_freqs * a * tau) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(a, 0, f, t) + 1] = (signed char)((((sin(2 * PI * rf_freqs * a * tau) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						// Y polarization
						data_sim[2 * data_in_idx(a, 1, f, t)] = (signed char)((((cos(2 * PI * rf_freqs * a * tau) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256);
						data_sim[2 * data_in_idx(a, 1, f, t) + 1] = (signed char)((((sin(2 * PI * rf_freqs * a * tau) - tmp_min)/(tmp_max-tmp_min)) - 0.5)*256); // Make this negative if a different polarization is tested
					}else{
						// X polarization
						data_sim[2 * data_in_idx(a, 0, f, t)] = 0;
						data_sim[2 * data_in_idx(a, 0, f, t) + 1] = 0;
						// Y polarization
						data_sim[2 * data_in_idx(a, 1, f, t)] = 0;
						data_sim[2 * data_in_idx(a, 1, f, t) + 1] = 0; // Make this negative if a different polarization is tested
					}
				}
			}
		}
	}
	//for (int a = 0; a < N_ANT; a++){
	//	printf("Antenna %d = %f\n", (a+1), data_sim[2 * data_in_idx(a, 0, 0, 0)]);
	//}
	return data_sim;
}

// Generate simulated weights or coefficients
float* simulate_coefficients() {
	float* coeff_sim;
	coeff_sim = (float*)calloc(N_COEFF, sizeof(float));
	//checkCuda(hipHostMalloc((void **)&coeff_sim, N_COEFF * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Scale each beam by incrementing value i.e. beam 1 = 1, beam 2 = 2, ..., beam 64 = 64
	sim_flag = 2 -> Scale each beam by incrementing value in a particular bin (bin 3 and 6 for now). Match simulated data sim_flag = 2
	sim flag = 3 -> Simulated beams from 58 to 122 degrees. Assuming a ULA.
	*/
	int sim_flag = 0;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_COEFF / 2); i++) {
			coeff_sim[2 * i] = 1;
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
		
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		//float chan_band = 1.59; // Fine channel bandwidth in Hz

		//float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		//for (int i = 0; i < N_FREQ; i++) {
		//	rf_freqs[i] = chan_band * i + c_freq;
		//}

		//float* theta = (float*)calloc(N_TIME, sizeof(float)); // Beam angle from 58 to 122 degrees
		//float* tau_beam = (float*)calloc(N_BEAM, sizeof(float)); // Delay

		float theta = 0; // Beam angle from 58 to 122 degrees
		float tau_beam = 0; // Delay

		for (int b = 0; b < N_BEAM; b++) {
			theta = ((b - (N_BEAM / 2)) + 90)*PI/180; // Beam angle from 58 to 122 degrees - Given SOI at 90 deg or moving across array, the beam with the most power is beamm 33
			tau_beam = d * cos(theta) / c; // Delay
			for (int a = 0; a < N_ANT; a++) {
				coeff_sim[2 * coeff_idx(a, b)] = cos(2 * PI * c_freq * a * tau_beam);
				coeff_sim[2 * coeff_idx(a, b) + 1] = sin(2 * PI * c_freq * a * tau_beam);
			}
		}
	}
	//for (int b = 0; b < N_BEAM; b++){
	//	printf("Beam %d = %f\n", (b+1), coeff_sim[2 * coeff_idx(0, b)]);
	//}

	return coeff_sim;
}

// Generate weights or coefficients with calculated delays (with delay polynomials (tau), coarse frequency channel (coarse_chan), and epoch (t))
float* generate_coefficients(float* tau, float coarse_chan, float t) {
	float* coefficients;
	coefficients = (float*)calloc(N_COEFF, sizeof(float));
	float delay_rate = 0;
	float delay_offset = 0;

	for (int b = 0; b < N_BEAM; b++) {
		for (int a = 0; a < N_ANT; a++) {
			delay_rate = tau[delay_idx(1,a,b)];
			delay_offset = tau[delay_idx(0,a,b)];
			coefficients[2 * coeff_idx(a, b)] = cos(2 * PI * coarse_chan * (t*delay_rate + delay_offset));
			coefficients[2 * coeff_idx(a, b) + 1] = sin(2 * PI * coarse_chan * (t*delay_rate + delay_offset));
		}
	}

	return coefficients;
}

// The input_data_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
//void input_data_pin(float * data_in_pin) {
void input_data_pin(signed char * data_in_pin) {
	checkCuda(hipHostRegister(data_in_pin, N_INPUT*sizeof(signed char), hipHostRegisterPortable));
}

// The data_coeff_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
void coeff_pin(float * data_coeff_pin) {
	checkCuda(hipHostRegister(data_coeff_pin, N_COEFF*sizeof(float), hipHostRegisterPortable));
}

// The output_data_pin() function uses hipHostRegister() to allocate the output host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
void output_data_pin(float * data_out_pin) {
	checkCuda(hipHostRegister(data_out_pin, N_BF_POW*sizeof(float), hipHostRegisterPortable));
}

// The coefficient_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
//void coefficient_pin(float * coeff_pin) {
//	checkCuda(hipHostRegister(coeff_pin, N_COEFF*sizeof(float), hipHostRegisterPortable));
//}

// Unregister host arrays from pinned memory
void unregister_data(void * data_unregister){
	checkCuda(hipHostUnregister(data_unregister));
}

// Free memory
void cohbfCleanup() {
	// Free up GPU memory at the end of a program
	//if (h_data != NULL) {
	//	hipHostFree(h_data);
	//}
	if (d_data_char != NULL) {
		hipFree(d_data_char);
	}
	if (d_data_float != NULL) {
		hipFree(d_data_float);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
	if (d_coeff != NULL) {
		hipFree(d_coeff);
	}
	//if (d_coeff_comp != NULL) {
	//	hipFree(d_coeff_comp);
	//}
	//if (d_coh_bf_out != NULL) {
	//	hipFree(d_coh_bf_out);
	//}
	if (d_coh_bf_pow != NULL) {
		hipFree(d_coh_bf_pow);
	}
}

//Comment out main() function when compiling for hpguppi
/*// <----Uncomment here if testing standalone code
// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main() {
	printf("Here!\n");

	// Allocate memory to all arrays used by run_beamformer() 
	init_beamformer();

	// Generate simulated data
	signed char* sim_data = simulate_data();
	// Register the array in pinned memory to speed HtoD mem copy
	input_data_pin(sim_data);

	// Generate simulated weights or coefficients
	float* sim_coefficients = simulate_coefficients();
	//printf("Here3!\n");
	// Register the array in pinned memory to speed HtoD mem copy
	coeff_pin(sim_coefficients);

	printf("real sim_data: %d and imag sim_data: %d\n", sim_data[10485768], sim_data[10485769]);
	//printf("real sim_coef: %f and imag sim_coef: %f\n", sim_coefficients[104], sim_coefficients[105]);

	printf("real sim_data2: %d and imag sim_data2: %d\n", sim_data[8388616], sim_data[8388617]);
	//printf("real sim_coef2: %f and imag sim_coef2: %f\n", sim_coefficients[106], sim_coefficients[107]);
	// Allocate memory for output array
	float* output_data;
	//output_data = (float*)calloc(N_BF_POW, sizeof(float));
	//output_data = (float*)calloc(N_OUTPUT, sizeof(float));
	//output_data_pin(output_data);

	printf("Here5!\n");

	float time_taken = 0;
	float bf_time = 0;
	int num_runs = 10;

	// Start timing beamformer computation //
	struct timespec tval_before, tval_after;

	for(int ii = 0; ii < num_runs; ii++){
		// Start timing beamformer computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_before);

		// Run beamformer 
		output_data = run_beamformer(sim_data, sim_coefficients);
		//run_beamformer(h_data, h_coeff, output_data);

		// Stop timing beamforming computation //
		clock_gettime(CLOCK_MONOTONIC, &tval_after);
		time_taken = (float)(tval_after.tv_sec - tval_before.tv_sec); //*1e6; // Time in seconds since epoch
		time_taken = time_taken + (float)(tval_after.tv_nsec - tval_before.tv_nsec)*1e-9; // Time in nanoseconds since 'tv_sec - start and end'
		bf_time += time_taken;
		//printf("Time taken: %f s\n", time_taken);
	}
	printf("Average delay calculation time: %f s\n", bf_time/num_runs);

	printf("Here6, Beamformer output: %f \n", output_data[0]);
	
	// Write data to text file for analysis
	char output_filename[128];

	printf("Here7!\n");

	//strcpy(output_filename, "C:\Users\ruzie\OneDrive\Desktop\Work\CUDA_code\output_d.txt");
	strcpy(output_filename, "output_d_cuda.txt");

	printf("Here8!\n");

	FILE* output_file;

	printf("Here9!\n");

	output_file = fopen(output_filename, "w");

	printf("Here10!\n");

	for (int ii = 0; ii < N_BF_POW; ii++) {
		//fprintf(output_file, "%c\n", output_data[ii]);
		fprintf(output_file, "%g\n", output_data[ii]);
	}

	printf("Here11!\n");

	fclose(output_file);

	printf("Closed output file.\n");

	//unregister_data((float *)sim_data);
	//free(sim_data);
	printf("After unregister.\n");	
	//free(sim_coefficients);
	printf("After freeing coefficients.\n");
	//free(output_data);	

	printf("Freed output array and unregistered arrays in pinned memory.\n");

	// Free up device memory
	//hipHostFree(h_data);
	//hipHostFree(h_coeff);
	cohbfCleanup();

	printf("Here11!\n");

	return 0;
}
*/ // <----Uncomment here if testing standalone code
