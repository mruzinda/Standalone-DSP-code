#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
//#include <unistd.h>
#include <hipblas.h>
#include <time.h>
//#include <sys/time.h>
#include <iostream>
#include <string.h>
//#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "coherent_beamformer.h"

using namespace std;

// Check for CUDA error
void checkCUDAerr();

// Generate simulated data
float* simulate_data();

// Generate simulated weights or coefficients
float* simulate_coefficients();

// Perform transpose on the data and convert to floats
__global__
void data_transpose(float* data_in, hipComplex* data_tra);

// Convert weights from float to hipComplex
__global__
void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex);

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, hipComplex* coeff, float* output_data);

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power);

// Check for CUDA error
inline void checkCUDAerr(int kernel_idx){
	kernel_idx = kernel_idx + 1;
	hipError_t err_code;
	err_code = hipGetLastError();
	if (err_code != hipSuccess) {
		if(kernel_idx == 0){
			printf("COH_BF: Input data hipMemcpy() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 1){
			printf("COH_BF: Coefficient hipMemcpy() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 2){
			printf("COH_BF: data_transpose() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 3){
			printf("COH_BF: beamformer_coefficient() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 4){
			printf("COH_BF: coherent_beamformer() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 5){
			printf("COH_BF: beamformer_power() Failed: %s\n", hipGetErrorString(err_code));
		}
		if(kernel_idx == 6){
			printf("COH_BF: Final hipMemcpy Failed: %s\n", hipGetErrorString(err_code));
		}
	}
}

//float* h_data = NULL;
float* d_data_float = NULL;
hipComplex* d_data_comp = NULL;
float* d_coeff_float = NULL;
hipComplex* d_coeff_comp = NULL;
float* d_coh_bf_out = NULL;
float* d_coh_bf_pow = NULL;
// Allocate memory to all arrays 
void init_beamformer(){
	//hipHostMalloc((void **)&h_data, N_INPUT*sizeof(float));
	// Allocate memory for input data float type
	hipMalloc((void **)&d_data_float, N_INPUT*sizeof(float));
	// Allocate memory for input data hipComplex type
	hipMalloc((void **)&d_data_comp, N_INPUT*sizeof(hipComplex)/2);
	// Allocate memory for coefficients float type
	hipMalloc((void **)&d_coeff_float, N_COEFF*sizeof(float));
	// Allocate memory for coefficients hipComplex type
	hipMalloc((void **)&d_coeff_comp, N_COEFF*sizeof(hipComplex)/2);
	// Allocate memory for coherent beamformer output
	hipMalloc((void **)&d_coh_bf_out, N_OUTPUT*sizeof(float));
	// Allocate memory for output power of coherent beamformer
	hipMalloc((void **)&d_coh_bf_pow, N_BF_POW*sizeof(float));
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(float* data_in, hipComplex* data_tra){
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
    
	// If the input data is not float, just multiply it by '1.0f' to convert it to a float
	data_tra[data_tr_idx(p, t, f, a)].x = data_in[2*data_in_idx(p, t, f, a)];
	data_tra[data_tr_idx(p, t, f, a)].y = data_in[2*data_in_idx(p, t, f, a) + 1];
}

// Convert weights from float to hipComplex
__global__
void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex){
	// Product of antenna and beam dimensions exceeds 1024 so beams
	// are blocks rather than threads to allow for increase in numbers  
	int a = threadIdx.x; // Antenna index
	int b = blockIdx.y;  // Beam index
	int f = blockIdx.x;  // Frequency bin index 
  
	coeff_complex[coeff_idx(a, b, f)].x = coeff_float[2*coeff_idx(a, b, f)];
	coeff_complex[coeff_idx(a, b, f)].y = coeff_float[2*coeff_idx(a, b, f) + 1];
}

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, hipComplex* coeff, float* output_data){
	int p = threadIdx.x; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index
  
	hipComplex bf_product;
  
	for(int a = 0; a < N_ANT; a++){ // Antenna index
		// Complex multiplication of data and coefficients
		bf_product.x = input_data[data_tr_idx(p, t, f, a)].x*coeff[coeff_idx(a, b, f)].x
						- input_data[data_tr_idx(p, t, f, a)].y*coeff[coeff_idx(a, b, f)].y;
		bf_product.y = input_data[data_tr_idx(p, t, f, a)].x*coeff[coeff_idx(a, b, f)].y
						+ input_data[data_tr_idx(p, t, f, a)].y*coeff[coeff_idx(a, b, f)].x;

		// Beamform (Sum all antennas)
		output_data[2*coh_bf_idx(t, f, b)] += bf_product.x;
		output_data[2*coh_bf_idx(t, f, b) + 1] += bf_product.y;
	}
}

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power){
	int b = threadIdx.x; // Beam index
	int f = blockIdx.x;  // Frequency bin index
	int t = blockIdx.y;  // Time sample index
  
	// Power = Absolute value squared of output -> r^2 + i^2
	bf_power[2*coh_bf_idx(t, f, b)] = (bf_volt[2*coh_bf_idx(t, f, b)]*bf_volt[2*coh_bf_idx(t, f, b)]) 
									+ (bf_volt[2*coh_bf_idx(t, f, b) + 1]*bf_volt[2*coh_bf_idx(t, f, b) + 1]);
}

// Run beamformer
void run_beamformer(float* data_in, float* h_coefficient, float* data_out){
	int kern_idx = 0; // Kernel index in run_beamformer function for printing error
  
	/*
	// Allocate input data in pinned memory 
	// (This may take longer than it's worth to implement pinned memory)
	*h_data = *data_in;
	*/
  
	// Transpose kernel: Specify grid and block dimensions
	dim3 dimBlock_transpose(N_ANT, N_POL, 1);
	dim3 dimGrid_transpose(N_BIN, N_TIME, 1);
  
	// Beamformer coefficient kernel (float to complex): Specify grid and block dimensions
	dim3 dimBlock_bf_coeff(N_ANT, 1, 1);
	dim3 dimGrid_bf_coeff(N_BIN, N_BEAM, 1);

	// Coherent beamformer kernel: Specify grid and block dimensions
	dim3 dimBlock_coh_bf(N_POL, 1, 1);
	dim3 dimGrid_coh_bf(N_BIN, N_TIME, N_BEAM);

	// Output power of beamformer kernel: Specify grid and block dimensions
	dim3 dimBlock_bf_pow(N_BEAM, 1, 1);
	dim3 dimGrid_bf_pow(N_BIN, N_TIME, 1);

	float* d_data_in = d_data_float;
	hipComplex* d_data_tra = d_data_comp;
	float* d_coeff_f = d_coeff_float;
	hipComplex* d_coeff_c = d_coeff_comp;
	float* d_bf_output = d_coh_bf_out;
	float* d_bf_pow = d_coh_bf_pow;
  
	// Copy input data from host to device
	hipMemcpy(d_data_in, data_in, N_INPUT*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAerr(kern_idx);
  
	// Copy beamformer coefficients from host to device
	hipMemcpy(d_coeff_f, h_coefficient, N_COEFF*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAerr(kern_idx);

	// Perform transpose on the data and convert to floats  
	data_transpose<<<dimGrid_transpose, dimBlock_transpose>>>(d_data_in, d_data_tra);
	checkCUDAerr(kern_idx);

	// Convert weights from float to hipComplex    
	beamformer_coefficient<<<dimGrid_bf_coeff, dimBlock_bf_coeff>>>(d_coeff_f, d_coeff_c);
	checkCUDAerr(kern_idx);
  
	// Perform beamforming operation
	coherent_beamformer<<<dimGrid_coh_bf, dimBlock_coh_bf>>>(d_data_tra, d_coeff_c, d_bf_output);
	checkCUDAerr(kern_idx);

	// Compute power of beamformer output (abs()^2)    
	beamformer_power<<<dimGrid_bf_pow, dimBlock_bf_pow>>>(d_bf_output, d_bf_pow);
	checkCUDAerr(kern_idx);
  
	// Copy output power from device to host
	hipMemcpy(data_out, d_bf_pow, N_BF_POW*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAerr(kern_idx);
  
	kern_idx = 0; // Reset the kernel index for the CUDA error check
	/*
	// Option to copy output power or voltage to host
	if(pow_flag == 0){
		hipMemcpy(data_out, d_bf_output, N_OUTPUT*sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAerr(kern_idx);  
	}else{
		beamformer_power<<<dimGrid_bf_pow, dimBlock_bf_pow>>>(d_bf_output, d_bf_pow);
		checkCUDAerr(kern_idx);
  
		hipMemcpy(data_out, d_bf_pow, N_BF_POW*sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAerr(kern_idx);
	}
	*/
}

// Generate simulated data
float* simulate_data(){
	float* data_sim;
	data_sim = (float*)calloc(N_INPUT, sizeof(float));
	for(int i = 0; i<(N_INPUT/2); i++){
		data_sim[2*i] = 1;
	}
	return data_sim;
}

// Generate simulated weights or coefficients
float* simulate_coefficients(){
	float* coeff_sim;
	
	printf("Here in sim coeff!\n");
	
	coeff_sim = (float*)calloc(N_COEFF, sizeof(float));
	
	printf("Here in sim coeff 1!\n");
	
	for(int i = 0; i<(N_COEFF/2); i++){
		coeff_sim[2*i] = 1;
		//printf("Here in sim coeff: %f, idx = %d\n", coeff_sim[2*i], i);
	}
	
	printf("Here in sim coeff 2!\n");
	
	return coeff_sim;
}

// Free memory
void cohbfCleanup() {
	// Free up GPU memory at the end of a program
	if (d_data_float != NULL) {
		hipFree(d_data_float);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
	if (d_coeff_float != NULL) {
		hipFree(d_coeff_float);
	}
	if (d_coeff_comp != NULL) {
		hipFree(d_coeff_comp);
	}
	if (d_coh_bf_out != NULL) {
		hipFree(d_coh_bf_out);
	}
	if (d_coh_bf_pow != NULL) {
		hipFree(d_coh_bf_pow);
	}
}

// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main(){
    printf("Here!\n");
	// Generate simulated data
	float* sim_data = simulate_data();
    
	printf("Here1!\n");
	
	// Generate simulated weights or coefficients
	float* sim_coefficients = simulate_coefficients();
    
	printf("Here2!\n");
	
	// Allocate memory to all arrays used by run_beamformer() 
	init_beamformer();
    
	printf("Here3!\n");
	
	// Allocate memory for output array
	float* output_data;
	output_data = (float*)calloc(N_BF_POW,sizeof(float));
    
	printf("Here4!\n");
	
	// Run beamformer 
	run_beamformer(sim_data, sim_coefficients, output_data);
    
	printf("Here5!\n");
	
	// Write data to text file for analysis
	char output_filename[128];
	
	printf("Here6!\n"); 
	
	//strcpy(output_filename, "C:\Users\ruzie\OneDrive\Desktop\Work\CUDA_code\output_d.txt");
	strcpy(output_filename, "output_d.txt");
	
	printf("Here7!\n");
	
	FILE* output_file;
	
	printf("Here8!\n");
	
	output_file = fopen(output_filename, "w");
	
	printf("Here9!\n");
	
	for(int ii = 0; ii<N_BF_POW; ii++){
		fprintf(output_file, "%g\n", output_data[ii]);
	}
	
	printf("Here10!\n");
	
	fclose(output_file);
	
	printf("Closed output file.\n");
	
	free(output_data);
	
	printf("Freed output array memory.\n");

	// Free up device memory
	cohbfCleanup();
	
	printf("Here11!\n");
    
	return 0;
}