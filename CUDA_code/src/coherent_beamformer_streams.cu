#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hiprand.h>
#include <assert.h>
//#include <unistd.h>
#include <hipblas.h>
#include <time.h>
//#include <sys/time.h>
#include <iostream>
#include <string.h>
//#include <complex.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "coherent_beamformer.h"

using namespace std;

// Generate simulated data
float* simulate_data();

// Generate simulated weights or coefficients
float* simulate_coefficients();

// Perform transpose on the data and convert to floats
__global__
void data_transpose(float* data_in, hipComplex* data_tra);

// Convert weights from float to hipComplex
//__global__
//void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex);

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data);

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power);

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
//#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

//float* h_data = NULL;
//float* h_coeff = NULL;
float* d_data_float = NULL;
hipComplex* d_data_comp = NULL;
float* d_coeff = NULL;
//hipComplex* d_coeff_comp = NULL;
//float* d_coh_bf_out = NULL;
float* d_coh_bf_pow = NULL;
// Allocate memory to all arrays 
void init_beamformer() {
	printf("Here In init_beamformer()! \n");
	// Allocate pinned memory for input data
	//checkCuda(hipHostMalloc((void **)&h_data, N_INPUT * sizeof(float)));
	//printf("Here 1st hipHostMalloc! \n");
	
	// Allocate pinnted memery for beamformer coefficients
	//checkCuda(hipHostMalloc((void **)&h_coeff, N_COEFF * sizeof(float)));
	//printf("Here 2nd hipHostMalloc! \n");
	//hipHostAlloc((void **)&h_data, N_INPUT * sizeof(float));

	// Allocate memory for input data float type
	checkCuda(hipMalloc((void **)&d_data_float, N_INPUT * sizeof(float)));
	printf("Here 1st hipMalloc! \n");

	// Allocate memory for input data hipComplex type
	checkCuda(hipMalloc((void **)&d_data_comp, N_INPUT * sizeof(hipComplex) / 2));
	printf("Here 2nd hipMalloc! \n");

	/*
	size_t f, t;
    	hipSetDevice(0);
    	hipMemGetInfo(&f, &t);
    	fprintf(stdout,"Free: %zu bytes, Available: %zu bytes \n",f,t);
	*/

	// Allocate memory for coefficients float type
	checkCuda(hipMalloc((void **)&d_coeff, N_COEFF * sizeof(float)));
	printf("Here 3rd hipMalloc! \n");

	// Allocate memory for coefficients hipComplex type
	//checkCuda(hipMalloc((void **)&d_coeff_comp, N_COEFF * sizeof(hipComplex) / 2));
	//printf("Here 4th hipMalloc! \n");

	// Allocate memory for coherent beamformer output
	//checkCuda(hipMalloc((void **)&d_coh_bf_out, N_OUTPUT * sizeof(float)));
	//printf("Here 5th hipMalloc! \n");

	// Allocate memory for output power of coherent beamformer
        checkCuda(hipMalloc((void **)&d_coh_bf_pow, N_BF_POW * sizeof(float)));
	printf("Here 4th hipMalloc! \n");

	return;
}

// Perform transpose on the data and convert to floats
__global__
void data_transpose(float* data_in, hipComplex* data_tra, unsigned int offset) {
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index

	// If the input data is not float e.g. signed char, just multiply it by '1.0f' to convert it to a float
	unsigned int h = offset + data_tr_idx(a, p, f, t);
	data_tra[h].x = data_in[2*h];
	data_tra[h].y = data_in[2*h + 1];
	
	return;
}

/*
// Convert weights from float to hipComplex
__global__
void beamformer_coefficient(float* coeff_float, hipComplex* coeff_complex) {
	// Product of antenna and beam dimensions exceeds 1024 so beams
	// are blocks rather than threads to allow for increase in numbers  
	int a = threadIdx.x; // Antenna index
	int p = threadIdx.y; // Polarization index
	int b = blockIdx.y;  // Beam index
	int f = blockIdx.x;  // Frequency bin index 
	coeff_complex[coeff_idx(a, p, b, f)].x = coeff_float[2*coeff_idx(a, p, b, f)];
	coeff_complex[coeff_idx(a, p, b, f)].y = coeff_float[2*coeff_idx(a, p, b, f) + 1];
	
	return;
}
*/

// Perform beamforming operation
__global__
void coherent_beamformer(hipComplex* input_data, float* coeff, float* output_data, unsigned int offset) {
	/*
	int p = threadIdx.x; // Polarization index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index
	hipComplex bf_product;
	hipComplex bf_in_data;
	hipComplex bf_coeff;
	for (int a = 0; a < N_ANT; a++) { // Antenna index
		bf_in_data.x = input_data[data_tr_idx(a, p, f, t)].x;
		bf_in_data.y = input_data[data_tr_idx(a, p, f, t)].y;
		bf_coeff.x = coeff[coeff_idx(a, p, b, f)].x;
		bf_coeff.y = coeff[coeff_idx(a, p, b, f)].y;
		// Complex multiplication of data and coefficients
		bf_product.x = (bf_in_data.x * bf_coeff.x) - (bf_in_data.y * bf_coeff.y);
		bf_product.y = (bf_in_data.x * bf_coeff.y) + (bf_in_data.y * bf_coeff.x);
		// Beamform (Sum all antennas)
		output_data[2*coh_bf_idx(p, b, f, t)] += bf_product.x;
		output_data[2*coh_bf_idx(p, b, f, t) + 1] += bf_product.y;
	}
	*/
	int a = threadIdx.x; // Antenna index
	int f = blockIdx.x;  // Frequency index
	int t = blockIdx.y;  // Time sample index
	int b = blockIdx.z;  // Beam index

	__shared__ hipFloatComplex reduced_mul[N_ANT];

	for (int p = 0; p < N_POL; p++) { // Polarization index
		// Reinitialize output_data since we are using the input data array to be more efficient
		unsigned int h = offset + coh_bf_idx(p, b, f, t);
		output_data[2 * h] = 0;
		output_data[2 * h + 1] = 0;


		unsigned int i = offset + data_tr_idx(a, p, f, t);
		unsigned int w = coeff_idx(a, b);

		if (a < N_ANT) {
			reduced_mul[a].x = input_data[i].x * coeff[2*w] + input_data[i].y * coeff[2*w + 1];
			reduced_mul[a].y = input_data[i].y * coeff[2*w] - input_data[i].x * coeff[2*w + 1];
		}
		else {
			reduced_mul[a].x = 0;
			reduced_mul[a].y = 0;
		}
		__syncthreads();

		for (int k = blockDim.x / 2; k > 0; k >>= 1) {
			if (a < k) {
				reduced_mul[a].x += reduced_mul[a + k].x;
				reduced_mul[a].y += reduced_mul[a + k].y;
			}
			__syncthreads();
		}
		if (a == 0) {
			unsigned int h = offset + coh_bf_idx(p, b, f, t);
			output_data[2 * h] += reduced_mul[0].x;
			output_data[2 * h + 1] += reduced_mul[0].y;
		}
		
	}
	return;
}

// Compute power of beamformer output (abs()^2)
__global__
void beamformer_power(float* bf_volt, float* bf_power, unsigned int offset) {
	int b = threadIdx.x; // Beam index
	int f = blockIdx.x;  // Frequency bin index
	int t = blockIdx.y;  // Time sample index

	// Power = Absolute value squared of output -> r^2 + i^2
	unsigned int xp = offset + coh_bf_idx(0, b, f, t); // X polarization
	unsigned int yp = offset + coh_bf_idx(1, b, f, t); // Y polarization
	
	float x_pol_pow = (bf_volt[2 * xp] * bf_volt[2 * xp]) + (bf_volt[2 * xp + 1] * bf_volt[2 * xp + 1]); // XX*
	float y_pol_pow = (bf_volt[2 * yp] * bf_volt[2 * yp]) + (bf_volt[2 * yp + 1] * bf_volt[2 * yp + 1]); // YY*

	bf_power[offset + pow_bf_idx(b, f, t)] = x_pol_pow + y_pol_pow; // XX* + YY*

	/*
	bf_power[pow_bf_idx(0, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*xp]) + (bf_volt[2*xp + 1]*bf_volt[2*xp + 1]); // XX*
	bf_power[pow_bf_idx(1, b, f, t)] = (bf_volt[2*yp]*bf_volt[2*yp]) + (bf_volt[2*yp + 1]*bf_volt[2*yp + 1]); // YY*
	bf_power[pow_bf_idx(2, b, f, t)] = (bf_volt[2*xp]*bf_volt[2*yp]) + (bf_volt[2*xp + 1]*bf_volt[2*yp + 1]); // XY* real
	bf_power[pow_bf_idx(3, b, f, t)] = (bf_volt[2*xp + 1]*bf_volt[2*yp]) - (bf_volt[2*xp]*bf_volt[2*yp + 1]); // XY* imag
	*/

	return;
}

// Run beamformer
void run_beamformer(float* data_in, float* h_coefficient, float* data_out) {
	/*
	// Allocate input data in pinned memory
	// (This may take longer than it's worth to implement pinned memory)
	*h_data = *data_in;
	*/

	hipError_t err_code;

	const unsigned int num_streams = 1; // Number of streams that make up all of the data. Split in frequency blocks (largest dimension)
	const unsigned int freq_chans = N_FREQ/num_streams;
	const int nStreams = num_streams; // Number of streams  
	printf("Total frequency channels: %u , num streams: %d \n", freq_chans, nStreams);

	// Transpose kernel: Specify grid and block dimensions
	dim3 dimBlock_transpose(N_ANT, N_POL, 1);
	dim3 dimGrid_transpose(freq_chans, N_TIME, 1);

	// Beamformer coefficient kernel (float to complex): Specify grid and block dimensions
	//dim3 dimBlock_bf_coeff(N_ANT, N_POL, 1);
	//dim3 dimGrid_bf_coeff(N_FREQ, N_BEAM, 1);

	// Coherent beamformer kernel: Specify grid and block dimensions
	//dim3 dimBlock_coh_bf(N_ANT, N_POL, 1);
	dim3 dimBlock_coh_bf(N_ANT, 1, 1);
	dim3 dimGrid_coh_bf(freq_chans, N_TIME, N_BEAM);

	// Output power of beamformer kernel: Specify grid and block dimensions
	dim3 dimBlock_bf_pow(N_BEAM, 1, 1);
	dim3 dimGrid_bf_pow(freq_chans, N_TIME, 1);

	float* d_data_in = d_data_float;
	hipComplex* d_data_tra = d_data_comp;
	float* d_coefficient = d_coeff;
	//hipComplex* d_coeff_c = d_coeff_comp;
	//float* d_bf_output = d_coh_bf_out;
	float* d_bf_pow = d_coh_bf_pow;
        
	printf("Before hipMemcpy(HtoD) coefficients! \n");
	// Copy beamformer coefficients from host to device
	checkCuda(hipMemcpy(d_coefficient, h_coefficient, N_COEFF * sizeof(float), hipMemcpyHostToDevice));
	printf("Here hipMemcpy(HtoD) coefficients! \n");

	// CUDA streams and events applied for optimization to possibly eliminate stalls.
	// hipMemcpy(HtoD) and data_restructure kernel	
	const unsigned int streamSizeIn = (unsigned int)(2*N_ANT*N_POL*N_TIME*freq_chans);
	const unsigned long int streamBytesIn = (unsigned long int)(streamSizeIn * sizeof(float));
	printf("Input size: %u in bytes: %lu \n", streamSizeIn, streamBytesIn);

	// hipMemcpy(HtoD) coefficients
	//const int streamSizeCo = N_ANT*N_BEAM*N_TIME/nStreams;
	//const int streamBytesCo = streamSizeCo * sizeof(float);

	// coherent_beamformer kernel  
	const unsigned int streamSizeBF = (unsigned int)(2*N_BEAM*N_POL*N_TIME*freq_chans);
	//const unsigned int streamBytesBF = streamSizeBF * sizeof(float);
	printf("BF output size: %u \n", streamSizeBF);

	// beamformer_power kernel and hipMemcpy(DtoH)
	const unsigned int streamSizePow = (unsigned int)(N_BEAM*N_TIME*freq_chans);
	const unsigned int streamBytesPow = streamSizePow * sizeof(float);	
	printf("BF power output size: %u in bytes: %u \n", streamSizePow, streamBytesPow);

	// Create events and streams
	// Events ////////////////////////////////////
	hipEvent_t startEvent, stopEvent;
	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));		
	checkCuda(hipEventRecord(startEvent, 0));
	/////////////////////////////////////////////

	hipStream_t stream[nStreams];

	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamCreate(&stream[i]));
	}

	for (int i = 0; i < nStreams; ++i){

		unsigned int offset_in = i * streamSizeIn;
		// Copy input data from host to device
		checkCuda(hipMemcpyAsync(&d_data_in[offset_in], &data_in[offset_in], streamBytesIn, hipMemcpyHostToDevice, stream[i]));
		printf("First hipMemcpy(HtoD) in run_beamformer() \n");

		// Perform transpose on the data and convert to floats  
		data_transpose<<<dimGrid_transpose, dimBlock_transpose, 0, stream[i]>>>(d_data_in, d_data_tra, offset_in);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: data_transpose() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		printf("Here data_transpose! \n");

		// Perform beamforming operation
		// Use d_data_in for output since it is no longer being utilized,
		// and it is the same size as the output (4 GiB).
		unsigned int offset_bf = i * streamSizeBF;
		coherent_beamformer<<<dimGrid_coh_bf, dimBlock_coh_bf, 0, stream[i]>>>(d_data_tra, d_coefficient, d_data_in, offset_bf);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: coherent_beamformer() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		printf("Here coherent_beamformer! \n");	

		// Compute power of beamformer output (abs()^2)
		unsigned int offset_pow = i * streamSizePow;
		beamformer_power<<<dimGrid_bf_pow, dimBlock_bf_pow, 0, stream[i]>>>(d_data_in, d_bf_pow, offset_pow);
		err_code = hipGetLastError();
		if (err_code != hipSuccess) {
			printf("BF: beamformer_power() kernel Failed: %s\n", hipGetErrorString(err_code));
		}
		printf("Here beamformer_power! \n");

		// Copy output power from device to host
		checkCuda(hipMemcpyAsync(&data_out[offset_pow], &d_bf_pow[offset_pow], streamBytesPow, hipMemcpyDeviceToHost, stream[i]));
		printf("Here hipMemcpy(DtoH)! \n");

	}

	// Events ////////////////////////////////////
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));
	/////////////////////////////////////////////
	////////////////////////////////////////////////////////////////////////////////
	// Clean up streams
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
	for (int i = 0; i < nStreams; ++i) {
		checkCuda(hipStreamDestroy(stream[i]));
	}

	return;
}

// Generate simulated data
float* simulate_data() {
	float* data_sim;
	data_sim = (float*)calloc(N_INPUT, sizeof(float));
	//checkCuda(hipHostMalloc((void **)&data_sim, N_INPUT * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Repeating sequence of 1 to 64
	sim_flag = 2 -> Sequence of 1 to 64 placed in a particular bin (bin 6 for now)
	sim flag = 3 -> Simulated radio source in center beam assuming ULA
	*/
	int sim_flag = 3;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_INPUT / 2); i++) {
			data_sim[2 * i] = 1;
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int f = 0; f < N_FREQ; f++) {
					for (int a = 0; a < N_ANT; a++) {
						if (tmp >= N_ANT) {
							tmp = 0;
						}
						tmp = (tmp + 1) % (N_ANT+1);
						data_sim[2 * data_in_idx(a, p, f, t)] = tmp;
					}
				}
			}
		}
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int p = 0; p < N_POL; p++) {
			for (int t = 0; t < N_TIME; t++) {
				for (int a = 0; a < N_ANT; a++) {
					if (tmp >= N_ANT) {
						tmp = 0;
					}
					tmp = (tmp + 1) % (N_ANT+1);
					data_sim[2 * data_in_idx(a, p, 5, t)] = tmp;
					data_sim[2 * data_in_idx(a, p, 2, t)] = tmp;
				}
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		float chan_band = 1; // Fine channel bandwidth in Hz

		float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		for (int i = 0; i < N_FREQ; i++) {
			rf_freqs[i] = chan_band * i + c_freq;
		}

		float* theta = (float*)calloc(N_TIME, sizeof(float)); // SOI direction/angle of arrival
		float* tau = (float*)calloc(N_TIME, sizeof(float)); // Delay

		for (int t = 0; t < N_TIME; t++) {
			theta[t] = (t - (N_TIME / 2)) + 90; // SOI direction/angle of arrival -> Moving across array over time i.e. angle changes each time sample
			tau[t] = d * cos(theta[t]) / c; // Delay
			for (int f = 0; f < N_FREQ; f++) {
				for (int a = 0; a < N_ANT; a++) {
					// X polarization
					data_sim[2 * data_in_idx(a, 0, f, t)] = cos(2 * PI * rf_freqs[f] * a * tau[t]);
					data_sim[2 * data_in_idx(a, 0, f, t) + 1] = sin(2 * PI * rf_freqs[f] * a * tau[t]);
					// Y polarization
					data_sim[2 * data_in_idx(a, 1, f, t)] = cos(2 * PI * rf_freqs[f] * a * tau[t]);
					data_sim[2 * data_in_idx(a, 1, f, t) + 1] = sin(2 * PI * rf_freqs[f] * a * tau[t]); // Make this negative if a different polarization is tested
				}
			}
		}
	}
	//for (int a = 0; a < N_ANT; a++){
	//	printf("Antenna %d = %f\n", (a+1), data_sim[2 * data_in_idx(a, 0, 0, 0)]);
	//}
	return data_sim;
}

// Generate simulated weights or coefficients
float* simulate_coefficients() {
	float* coeff_sim;
	coeff_sim = (float*)calloc(N_COEFF, sizeof(float));
	//checkCuda(hipHostMalloc((void **)&coeff_sim, N_COEFF * sizeof(float)));

	/*
	'sim_flag' is a flag that indicates the kind of data that is simulated.
	sim_flag = 0 -> Ones
	sim_flag = 1 -> Scale each beam by incrementing value i.e. beam 1 = 1, beam 2 = 2, ..., beam 64 = 64
	sim_flag = 2 -> Scale each beam by incrementing value in a particular bin (bin 3 and 6 for now). Match simulated data sim_flag = 2
	sim flag = 3 -> Simulated beams from 58 to 122 degrees. Assuming a ULA.
	*/
	int sim_flag = 3;
	if (sim_flag == 0) {
		for (int i = 0; i < (N_COEFF / 2); i++) {
			coeff_sim[2 * i] = 1;
		}
	}
	if (sim_flag == 1) {
		int tmp = 0;
		
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
		
	}
	if (sim_flag == 2) {
		int tmp = 0;
		for (int a = 0; a < N_ANT; a++) {
			for (int b = 0; b < N_BEAM; b++) {
				if (tmp >= N_BEAM) {
					tmp = 0;
				}
				tmp = (tmp + 1) % (N_BEAM + 1);
				coeff_sim[2 * coeff_idx(a, b)] = tmp;
			}
		}
	}
	if (sim_flag == 3) {
		float c = 3e8; // Speed of light
		float c_freq = 1.25e9; // Center frequency
		float lambda = c / c_freq; // Wavelength
		float d = lambda / 2; // Distance between antennas
		//float chan_band = 1.59; // Fine channel bandwidth in Hz

		//float* rf_freqs = (float*)calloc(N_FREQ, sizeof(float));
		//for (int i = 0; i < N_FREQ; i++) {
		//	rf_freqs[i] = chan_band * i + c_freq;
		//}

		float* theta = (float*)calloc(N_TIME, sizeof(float)); // Beam angle from 58 to 122 degrees
		float* tau_beam = (float*)calloc(N_BEAM, sizeof(float)); // Delay

		for (int b = 0; b < N_BEAM; b++) {
			theta[b] = (b - (N_BEAM / 2)) + 90; // Beam angle from 58 to 122 degrees - Given SOI at 90 deg or moving across array, the beam with the most power is beamm 33
			tau_beam[b] = d * cos(theta[b]) / c; // Delay
			for (int a = 0; a < N_ANT; a++) {
				coeff_sim[2 * coeff_idx(a, b)] = cos(2 * PI * c_freq * a * tau_beam[b]);
				coeff_sim[2 * coeff_idx(a, b) + 1] = sin(2 * PI * c_freq * a * tau_beam[b]);
			}
		}
	}
	//for (int b = 0; b < N_BEAM; b++){
	//	printf("Beam %d = %f\n", (b+1), coeff_sim[2 * coeff_idx(0, b)]);
	//}

	return coeff_sim;
}

// The input_data_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
void input_data_pin(float * data_pin) {
	checkCuda(hipHostRegister(data_pin, N_INPUT*sizeof(float), hipHostRegisterPortable));
}

// The coefficient_pin() function uses hipHostRegister() to allocate the input host
// array in pinned memory.
// This speeds up the hipMemcpy() and enables implementation into HASHPIPE/RTOS.
//void coefficient_pin(float * coeff_pin) {
//	checkCuda(hipHostRegister(coeff_pin, N_COEFF*sizeof(float), hipHostRegisterPortable));
//}

// Unregister host arrays from pinned memory
void unregister_data(void * data_unregister){
	checkCuda(hipHostUnregister(data_unregister));
}

// Free memory
void cohbfCleanup() {
	// Free up GPU memory at the end of a program
	//if (h_data != NULL) {
	//	hipHostFree(h_data);
	//}
	if (d_data_float != NULL) {
		hipFree(d_data_float);
	}
	if (d_data_comp != NULL) {
		hipFree(d_data_comp);
	}
	if (d_coeff != NULL) {
		hipFree(d_coeff);
	}
	//if (d_coeff_comp != NULL) {
	//	hipFree(d_coeff_comp);
	//}
	//if (d_coh_bf_out != NULL) {
	//	hipFree(d_coh_bf_out);
	//}
	if (d_coh_bf_pow != NULL) {
		hipFree(d_coh_bf_pow);
	}
}


// Test all of the kernels and functions, and write the output to
// a text file for analysis
int main() {
	printf("Here!\n");

	// Allocate memory to all arrays used by run_beamformer() 
	init_beamformer();


	printf("Here1!\n");
	// Generate simulated data
	float* sim_data = simulate_data();
	// Register the array in pinned memory to speed HtoD mem copy
	input_data_pin(sim_data);

	printf("Here2!\n");

	// Generate simulated weights or coefficients
	float* sim_coefficients = simulate_coefficients();
	//printf("Here3!\n");
	// Register the array in pinned memory to speed HtoD mem copy
	//coefficient_pin(sim_coefficients);

	printf("Here4!\n");

	// Allocate memory for output array
	float* output_data;
	output_data = (float*)calloc(N_BF_POW, sizeof(float));
	//output_data = (float*)calloc(N_OUTPUT, sizeof(float));

	printf("Here5!\n");

	// Run beamformer 
	run_beamformer(sim_data, sim_coefficients, output_data);
	//run_beamformer(h_data, h_coeff, output_data);

	printf("Here6, Beamformer output: %f \n", output_data[0]);
	
	// Write data to text file for analysis
	char output_filename[128];

	printf("Here7!\n");

	//strcpy(output_filename, "C:\Users\ruzie\OneDrive\Desktop\Work\CUDA_code\output_d.txt");
	strcpy(output_filename, "output_d_cuda.txt");

	printf("Here8!\n");

	FILE* output_file;

	printf("Here9!\n");

	output_file = fopen(output_filename, "w");

	printf("Here10!\n");

	for (int ii = 0; ii < N_BF_POW; ii++) {
		fprintf(output_file, "%g\n", output_data[ii]);
	}

	printf("Here11!\n");

	fclose(output_file);

	printf("Closed output file.\n");

	//unregister_data((float *)sim_data);
	free(sim_data);
	printf("After unregister.\n");	
	free(sim_coefficients);
	printf("After freeing coefficients.\n");
	free(output_data);	

	printf("Freed output array and unregistered arrays in pinned memory.\n");

	// Free up device memory
	//hipHostFree(h_data);
	//hipHostFree(h_coeff);
	cohbfCleanup();

	printf("Here11!\n");

	return 0;
}
